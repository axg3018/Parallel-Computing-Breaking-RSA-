
#include <hip/hip_runtime.h>
#define NT 1024

// Overall counter variables in global memory.

__device__ int z;
__device__ int m[3];




extern "C" __global__ void modCube (int c, int N){

	//Variable declarations
    int thr, size, rank;
    

    //Rank and size computations
    thr = threadIdx.x;
    size = gridDim.x*NT;
    rank = blockIdx.x*NT + thr;

    for (int x = rank; x < N; x += size){

        if (((((x*x)%N)*x)%N) == c){

        	m[z] = x;
        	atomicAdd (&z, 1);
    		
        }
        
    }			

    __syncthreads();


}